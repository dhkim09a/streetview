#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <algorithm>
#include <vector>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "db.h"
#include "search.h"

#ifdef PROFILE_CUDA
#define PROFILE_ON
#endif
#include "profile.h"

/* Use device 0 */
#define DEV_ID 0

typedef struct _ipoint_essence_t {
	float vec[VEC_DIM] __attribute__((aligned (4)));
} __attribute__((packed)) ipoint_essence_t;

/* FIXME: The result is a bit different from CPU's */
__global__ void doSearchKernel (int shared_mem_size, int needle_idx,
		ipoint_essence_t *needle, int needle_size,
		ipoint_t *haystack, int haystack_size,
		struct _interim *interim, int interim_size_local)
{
	if (threadIdx.x + needle_idx >= needle_size)
		return;

	register float dist;//, temp;
	int i, j, k;
	int batch;

	struct _interim *interim_local =
		&(interim[(interim_size_local * blockIdx.x)
				+ threadIdx.x + needle_idx]);
	batch = haystack_size / gridDim.x;
	int haystack_size_local = ((blockIdx.x + 1) * batch) > haystack_size ?
		(haystack_size % batch) : batch;

	/* Copy needle into local memory */
	ipoint_essence_t needle_local;
	for (i = 0; i < VEC_DIM; i++)
		needle_local.vec[i] = needle[threadIdx.x + needle_idx].vec[i];

	struct _interim interim_temp;
	if (interim_local->lat_first == 0) {
		interim_temp.dist_first = FLT_MAX;
		interim_temp.dist_second = FLT_MAX;
	}
	else {
		interim_temp.dist_first = interim_local->dist_first;
		interim_temp.dist_second = interim_local->dist_second;
		interim_temp.lat_first = interim_local->lat_first;
		interim_temp.lng_first = interim_local->lng_first;
	}

	extern __shared__ ipoint_t haystack_shared[];
	batch = shared_mem_size / sizeof(ipoint_t);
	int iter;
	for (k = 0; k <= (haystack_size_local / batch); k++) {

		iter = ((k + 1) * batch) > haystack_size_local ?
			(haystack_size_local % batch) : batch;

		/* Copy haystack into shared memory */
		if (threadIdx.x == 0)
			for (i = 0; i < iter; i++)
				haystack_shared[i] =
					haystack[((haystack_size / gridDim.x) * blockIdx.x)
					+ (k * batch) + i];

		__syncthreads();

		for (i = 0; i < iter; i++) {
			dist = 0;
#if REG >= 128
			register float hss[0x10];
			for (j = 0; j < VEC_DIM; j += 0x10)
#else
			register float hss[0x8];
			for (j = 0; j < VEC_DIM; j += 0x8)
#endif
			{
				hss[0x0] = haystack_shared[i].vec[j];
				hss[0x1] = haystack_shared[i].vec[j + 0x1];
				hss[0x2] = haystack_shared[i].vec[j + 0x2];
				hss[0x3] = haystack_shared[i].vec[j + 0x3];
				hss[0x4] = haystack_shared[i].vec[j + 0x4];
				hss[0x5] = haystack_shared[i].vec[j + 0x5];
				hss[0x6] = haystack_shared[i].vec[j + 0x6];
				hss[0x7] = haystack_shared[i].vec[j + 0x7];
#if REG >= 128
				hss[0x8] = haystack_shared[i].vec[j + 0x8];
				hss[0x9] = haystack_shared[i].vec[j + 0x9];
				hss[0xA] = haystack_shared[i].vec[j + 0xA];
				hss[0xB] = haystack_shared[i].vec[j + 0xB];
				hss[0xC] = haystack_shared[i].vec[j + 0xC];
				hss[0xD] = haystack_shared[i].vec[j + 0xD];
				hss[0xE] = haystack_shared[i].vec[j + 0xE];
				hss[0xF] = haystack_shared[i].vec[j + 0xF];
#endif
				dist
					+= ((needle_local.vec[j] - hss[0x0])
							* (needle_local.vec[j] - hss[0x0]))
					+ ((needle_local.vec[j + 0x1] - hss[0x1])
							* (needle_local.vec[j + 0x1] - hss[0x1]))
					+ ((needle_local.vec[j + 0x2] - hss[0x2])
							* (needle_local.vec[j + 0x2] - hss[0x2]))
					+ ((needle_local.vec[j + 0x3] - hss[0x3])
							* (needle_local.vec[j + 0x3] - hss[0x3]))
					+ ((needle_local.vec[j + 0x4] - hss[0x4])
							* (needle_local.vec[j + 0x4] - hss[0x4]))
					+ ((needle_local.vec[j + 0x5] - hss[0x5])
							* (needle_local.vec[j + 0x5] - hss[0x5]))
					+ ((needle_local.vec[j + 0x6] - hss[0x6])
							* (needle_local.vec[j + 0x6] - hss[0x6]))
					+ ((needle_local.vec[j + 0x7] - hss[0x7])
							* (needle_local.vec[j + 0x7] - hss[0x7]));
#if REG >= 128
					+ ((needle_local.vec[j + 0x8] - hss[0x8])
							* (needle_local.vec[j + 0x8] - hss[0x8]))
					+ ((needle_local.vec[j + 0x9] - hss[0x9])
							* (needle_local.vec[j + 0x9] - hss[0x9]))
					+ ((needle_local.vec[j + 0xA] - hss[0xA])
							* (needle_local.vec[j + 0xA] - hss[0xA]))
					+ ((needle_local.vec[j + 0xB] - hss[0xB])
							* (needle_local.vec[j + 0xB] - hss[0xB]))
					+ ((needle_local.vec[j + 0xC] - hss[0xC])
							* (needle_local.vec[j + 0xC] - hss[0xC]))
					+ ((needle_local.vec[j + 0xD] - hss[0xD])
							* (needle_local.vec[j + 0xD] - hss[0xD]))
					+ ((needle_local.vec[j = 0xE] - hss[0xE])
							* (needle_local.vec[j + 0xE] - hss[0xE]))
					+ ((needle_local.vec[j + 0xF] - hss[0xF])
							* (needle_local.vec[j + 0xF] - hss[0xF]));
#endif
			}
			if (dist < interim_temp.dist_first) {
				interim_temp.lat_first =
					haystack_shared[i].latitude;
				interim_temp.lng_first =
					haystack_shared[i].longitude;
				interim_temp.dist_second =
					interim_temp.dist_first;
				interim_temp.dist_first = dist;
			}
			else if (dist < interim_temp.dist_second)
				interim_temp.dist_second = dist;
		}
	}

	interim_local->lat_first = interim_temp.lat_first;
	interim_local->lng_first = interim_temp.lng_first;
	interim_local->dist_first = interim_temp.dist_first;
	interim_local->dist_second = interim_temp.dist_second;

	return;
}

int searchGPU (IpVec needle, ipoint_t *haystack, int haystack_size,
		struct _interim *result, int result_size, int dummy)
{
	hipSetDevice(DEV_ID);
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

	PROFILE_START();
	PROFILE_VAR(init_device);
	PROFILE_VAR(copy_needle);
	PROFILE_VAR(copy_haystack);
	PROFILE_VAR(run_kernel);
	PROFILE_VAR(copy_result);
	PROFILE_VAR(post_processing);

	int i, j, iter;
	float dist;

	ipoint_essence_t *needle_essence_h, *needle_essence_d;
	ipoint_t *haystack_d;
	struct _interim *interim_h, *interim_d;
	int needle_size = needle.size();
	hipError_t err;

	PROFILE_FROM(init_device);
#ifdef PROFILE_CUDA
	hipDeviceSynchronize();
#endif
	PROFILE_TO(init_device);

	hipDeviceProp_t device_prop;
	hipGetDeviceProperties(&device_prop, DEV_ID);

	hipStream_t *stream;

	unsigned int stream_dim = (unsigned int)device_prop.multiProcessorCount;
	unsigned int grid_dim = 1;
	unsigned int block_dim =
		MIN(needle_size, (unsigned int)device_prop.maxThreadsPerBlock);
	block_dim = MIN(block_dim, (unsigned int)(device_prop.regsPerBlock / REG));

	stream = (hipStream_t *)malloc(stream_dim * sizeof(hipStream_t));
	for (i = 0; i < (int)stream_dim; i++)
		hipStreamCreate(&stream[i]);

	needle_essence_h = (ipoint_essence_t *)malloc(
			needle_size * sizeof(ipoint_essence_t));
	for (i = 0; i < needle_size; i++)
		for (j = 0; j < VEC_DIM; j++)
			needle_essence_h[i].vec[j] = needle[i].descriptor[j];

	PROFILE_FROM(copy_needle);
	/* Copy needle to device */
	if (hipMalloc((void **)&needle_essence_d,
				needle_size * sizeof(ipoint_essence_t)) != hipSuccess) {
		fprintf(stderr, "hipMalloc(needle_essence_d) failed\n");
		return -1;
	}
	if (hipMemcpy(needle_essence_d, needle_essence_h,
				needle_size * sizeof(ipoint_essence_t),
				hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr,
				"hipMemcpy(needle_essence_d, needle_essence_h) failed\n");
		return -1;
	}
#ifdef PROFILE_CUDA
	hipDeviceSynchronize();
#endif
	PROFILE_TO(copy_needle);

	PROFILE_FROM(copy_haystack);
	/* Copy haystack to device */
	if (hipMalloc((void **)&haystack_d,
				haystack_size * sizeof(ipoint_t)) != hipSuccess) {
		fprintf(stderr, "hipMalloc(haystack_d) failed\n");
		return -1;
	}
#ifdef PROFILE_CUDA
	hipDeviceSynchronize();
#endif
	PROFILE_TO(copy_haystack);

	/* Allocate memory for result
	 * TODO: Still the result must be copied from device is about
	 * hundreads of MB. Need to reduce them. */
	if (hipMalloc((void **)&interim_d,
			grid_dim * stream_dim * sizeof(struct _interim) * needle_size) != hipSuccess) {
		fprintf(stderr, "hipMalloc(interim_d) failed\n");
		return -1;
	}
	if (hipMemset(interim_d, 0,
				grid_dim * stream_dim * sizeof(struct _interim) * needle_size) != hipSuccess) {
		fprintf(stderr, "hipMemset(interim_d) failed\n");
		return -1;
	}
	interim_h = (struct _interim *)malloc(
			grid_dim * stream_dim * sizeof(struct _interim) * needle_size);

	int stream_haystack_quota = haystack_size / stream_dim;
	int stream_haystack_size;
	for (j = 0; j < (int)stream_dim; j++) {
		stream_haystack_size
			= (j + 1) * stream_haystack_quota > haystack_size ?
			(haystack_size % stream_haystack_quota) : stream_haystack_quota;

		if (hipMemcpyAsync(
					(ipoint_t *)(&haystack_d[stream_haystack_quota * j]),
					(ipoint_t *)(&haystack[stream_haystack_quota * j]),
					stream_haystack_size * sizeof(ipoint_t),
					hipMemcpyHostToDevice, stream[j]) != hipSuccess) {
			fprintf(stderr, "hipMemcpy(haystack_d, haystack) failed\n");
			return -1;
		}
	}
	for (i = 0; i <= needle_size / block_dim; i++) {

		PROFILE_FROM(run_kernel);
		/* Run CUDA kernel */
		for (j = 0; j < (int)stream_dim; j++) {
		stream_haystack_size
			= (j + 1) * stream_haystack_quota > haystack_size ?
			(haystack_size % stream_haystack_quota) : stream_haystack_quota;

			doSearchKernel <<<
				grid_dim,
				(block_dim * (i + 1)) > needle_size ?
					(needle_size % block_dim) : block_dim,
				device_prop.sharedMemPerBlock,
				stream[j] >>>
					(device_prop.sharedMemPerBlock, i * block_dim,
					 needle_essence_d, needle_size,
					 (ipoint_t *)(&haystack_d[stream_haystack_quota * j]),
					 stream_haystack_size,
					 &interim_d[needle_size * j], needle_size);
		}
#ifdef PROFILE_CUDA
		hipDeviceSynchronize();
#endif
		PROFILE_TO(run_kernel);

	}

	PROFILE_FROM(copy_result);
	/* Copy result to host */
	err = hipMemcpy(interim_h, interim_d,
			grid_dim * stream_dim * sizeof(struct _interim) * needle_size,
			hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "hipMemcpy(interim_h, interim_d): %s\n",
				hipGetErrorString(err));
		return -1;
	}
#ifdef PROFILE_CUDA
	hipDeviceSynchronize();
#endif
	PROFILE_TO(copy_result);

	PROFILE_FROM(post_processing);
	iter = MIN((int)needle.size(), result_size);
	for (i = 0; i < iter; i++) {
		for (j = 0; j < (int)(grid_dim * stream_dim); j++) {
			if (result[i].dist_first == FLT_MAX) {
				result[i].lat_first =
					interim_h[(j * needle_size) + i].lat_first;
				result[i].lng_first =
					interim_h[(j * needle_size) + i].lng_first;
				result[i].dist_first =
					interim_h[(j * needle_size) + i].dist_first;
				result[i].dist_second =
					interim_h[(j * needle_size) + i].dist_second;
				continue;
			}

			dist = interim_h[(j * needle_size) + i].dist_first;
			if (dist < result[i].dist_first) {
				result[i].lat_first =
					interim_h[(j * needle_size) + i].lat_first;
				result[i].lng_first =
					interim_h[(j * needle_size) + i].lng_first;
				result[i].dist_second = result[i].dist_first;
				result[i].dist_first = dist;
			}
			else if (dist < result[i].dist_second)
				result[i].dist_second = dist;

			dist = interim_h[(j * needle_size) + i].dist_second;
			if (dist < result[i].dist_first) {
				result[i].lat_first =
					interim_h[(j * needle_size) + i].lat_first;
				result[i].lng_first =
					interim_h[(j * needle_size) + i].lng_first;
				result[i].dist_second = result[i].dist_first;
				result[i].dist_first = dist;
			}
			else if (dist < result[i].dist_second)
				result[i].dist_second = dist;
		}
	}
	PROFILE_TO(post_processing);

	free(needle_essence_h);
	free(interim_h);

	hipFree(needle_essence_d);
	hipFree(haystack_d);
	hipFree(interim_d);

	PROFILE_END();
	PROFILE_PRINT(stdout);
	
	return 0;
}
